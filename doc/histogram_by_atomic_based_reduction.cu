
#include <hip/hip_runtime.h>
__global__ void kernel_getHist(unsigned char* array, long size, 
                               unsigned int* histo, int buckets)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid>=size)   return;

    unsigned char value = array[tid];

    int bin = value % buckets;

    atomicAdd(&histo[bin],1);
}

void getHist(unsigned char* array, long size, unsigned int* histo,
             int buckets)
{
    unsigned char* dArray;
    hipMalloc(&dArray,size);
    hipMemcpy(dArray,array,size,hipMemcpyHostToDevice);

    unsigned int* dHist;
    hipMalloc(&dHist,buckets * sizeof(int));
    hipMemset(dHist,0,buckets * sizeof(int));

    dim3 block(32);
    dim3 grid((size + block.x - 1)/block.x);

    kernel_getHist<<<grid,block>>>(dArray,size,dHist,buckets);

    hipMemcpy(histo,dHist,buckets * sizeof(int),hipMemcpyDeviceToHost);

    hipFree(dArray);
    hipFree(dHist);
}